#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <vector>

#define MAX_THREADS 256
#define SIZE 524288

#define __START__ hipEventCreate(&start); hipEventCreate(&stop); hipEventRecord(start, 0);
#define __STOP__(_V) hipEventRecord(stop, 0); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop); _V.push_back(time); hipEventDestroy(start); hipEventDestroy(stop);
#define __NEXT__(_V) __STOP__(_V) __START__
#define __START_BIS__ hipEventCreate(&startBis); hipEventCreate(&stopBis); hipEventRecord(startBis, 0);
#define __STOP_BIS__(_V) hipEventRecord(stopBis, 0); hipEventSynchronize(stopBis); hipEventElapsedTime(&time, startBis, stopBis); _V.push_back(time); hipEventDestroy(startBis); hipEventDestroy(stopBis);
#define __NEXT_BIS__(_V) __STOP_BIS__(_V) __START_BIS__

__global__ void sumall_kernel(float *d_vector, float *d_result)
{
    __shared__ float cache[MAX_THREADS];

    int cacheIdx = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    cache[cacheIdx] = i >= SIZE ? 0. : d_vector[i];
    __syncthreads();

    if (i >= SIZE)
        return;

    int padding = blockDim.x/2;
    while (padding != 0)
    {
        if (cacheIdx < padding)
            cache[cacheIdx] += cache[cacheIdx + padding];

        __syncthreads();
        padding /= 2;
    }

    if (cacheIdx == 0)
        atomicAdd(&d_result[0], cache[0]);
}

void showMean(std::vector<float> v)
{
    float sum(0);
    for (unsigned int i(0) ; i!=v.size() ; i++)
        sum += v[i];
    std::cout << 1000.*sum/v.size() << " milliseconds" << std::endl;
}

int main(int argc, char **argv)
{
    std::cout << "SIZE (vs Reduce-Native): " << SIZE << std::endl;
    hipEvent_t start, stop, startBis, stopBis;
    std::vector<float> cudaRun, cudaKRun, nativeRun;
    float time(0);
    
    hipFree(0); // Force runtime API context establishment
    float h_vector[SIZE]; // For input and output
    for (unsigned int i(0) ; i!=SIZE ; i++)
        h_vector[i] = i;
    float h_result;
    
    for (unsigned int i(0) ; i!=1000 ; i++)
    {
        float *d_vector, *d_result;
        
        __START__
        hipMalloc(&d_vector, SIZE*sizeof(float));
        hipMalloc(&d_result, sizeof(float));
        hipMemcpy(d_vector, h_vector, SIZE*sizeof(float), hipMemcpyHostToDevice);
        hipMemset(d_result, 0, sizeof(float));
        __START_BIS__
        sumall_kernel<<<(SIZE+MAX_THREADS-1)/MAX_THREADS, MAX_THREADS>>>(d_vector, d_result);
        hipDeviceSynchronize(); // Block until the device is finished
        __STOP_BIS__(cudaKRun);
        hipMemcpy(h_vector, d_vector, SIZE*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_vector);
        hipFree(d_result);
        __NEXT__(cudaRun)
        
        h_result = 0.;
        for (unsigned int i(0) ; i!=SIZE ; i++)
            h_result += h_vector[i];
        __STOP__(nativeRun)
    }
    showMean(cudaRun);
    showMean(cudaKRun);
    showMean(nativeRun);
}    

