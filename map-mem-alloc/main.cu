#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <vector>

#define MAX_THREADS 256
#define SIZE 524288

#define __START__ hipEventCreate(&start); hipEventCreate(&stop); hipEventRecord(start, 0);
#define __STOP__(_V) hipEventRecord(stop, 0); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop); _V.push_back(time); hipEventDestroy(start); hipEventDestroy(stop);
#define __NEXT__(_V) __STOP__(_V) __START__

__global__ void square_kernel(float *d_vector)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= SIZE)
        return;
    d_vector[i] = d_vector[i]*d_vector[i];
}

void showMean(std::vector<float> v)
{
    float sum(0);
    for (unsigned int i(0) ; i!=v.size() ; i++)
        sum += v[i];
    std::cout << 1000.*sum/v.size() << " milliseconds" << std::endl;
}

int main(int argc, char **argv)
{
    std::cout << "SIZE (Memory Allocation): " << SIZE << std::endl;
    hipEvent_t start, stop;
    std::vector<float> cMalloc, cMemcpy1, cKernel, cMemcpy2, cFree;
    float time(0);
    
    hipFree(0); // Force runtime API context establishment
    float h_vector[SIZE]; // For input and output
    for (unsigned int i(0) ; i!=SIZE ; i++)
        h_vector[i] = i;
    
    for (unsigned int i(0) ; i!=1000 ; i++)
    {
        float *d_vector;
        __START__
        hipMalloc(&d_vector, SIZE*sizeof(float));
        __NEXT__(cMalloc);
        hipMemcpy(d_vector, h_vector, SIZE*sizeof(float), hipMemcpyHostToDevice);
        __NEXT__(cMemcpy1);
        square_kernel<<<(SIZE+MAX_THREADS-1)/MAX_THREADS, MAX_THREADS>>>(d_vector);
        hipDeviceSynchronize(); // Block until the device is finished
        __NEXT__(cKernel);
        hipMemcpy(h_vector, d_vector, SIZE*sizeof(float), hipMemcpyDeviceToHost);
        __NEXT__(cMemcpy2);
        hipFree(d_vector);
        __STOP__(cFree);
    }
    showMean(cMalloc);
    showMean(cMemcpy1);
    showMean(cKernel);
    showMean(cMemcpy2);
    showMean(cFree);
}    

