#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <vector>

#define MAX_THREADS 256
#define SIZE 524288

#define __START__ hipEventCreate(&start); hipEventCreate(&stop); hipEventRecord(start, 0);
#define __STOP__(_V) hipEventRecord(stop, 0); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop); _V.push_back(time); hipEventDestroy(start); hipEventDestroy(stop);
#define __NEXT__(_V) __STOP__(_V) __START__
#define __START_BIS__ hipEventCreate(&startBis); hipEventCreate(&stopBis); hipEventRecord(startBis, 0);
#define __STOP_BIS__(_V) hipEventRecord(stopBis, 0); hipEventSynchronize(stopBis); hipEventElapsedTime(&time, startBis, stopBis); _V.push_back(time); hipEventDestroy(startBis); hipEventDestroy(stopBis);
#define __NEXT_BIS__(_V) __STOP_BIS__(_V) __START_BIS__

__global__ void square_kernel(float *d_vector)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= SIZE)
        return;
    d_vector[i] = d_vector[i]*d_vector[i];
}

void showMean(std::vector<float> v)
{
    float sum(0);
    for (unsigned int i(0) ; i!=v.size() ; i++)
        sum += v[i];
    std::cout << 1000.*sum/v.size() << " microseconds" << std::endl;
}

int main(int argc, char **argv)
{
    std::cout << "SIZE (vs. Native): " << SIZE << std::endl;
    hipEvent_t start, stop, startBis, stopBis;
    std::vector<float> cRun, ckRun, hRun;
    float time(0);
    
    hipFree(0); // Force runtime API context establishment
    float h_vector[SIZE]; // For input and output
    for (unsigned int i(0) ; i!=SIZE ; i++)
        h_vector[i] = i;
    
    for (unsigned int i(0) ; i!=1000 ; i++)
    {
        float *d_vector;
        __START__
        hipMalloc(&d_vector, SIZE*sizeof(float));
        hipMemcpy(d_vector, h_vector, SIZE*sizeof(float), hipMemcpyHostToDevice);
        __START_BIS__
        square_kernel<<<(SIZE+MAX_THREADS-1)/MAX_THREADS, MAX_THREADS>>>(d_vector);
        hipDeviceSynchronize(); // Block until the device is finished
        __STOP_BIS__(ckRun)
        hipMemcpy(h_vector, d_vector, SIZE*sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_vector);
        __NEXT__(cRun)
        for (unsigned int i(0) ; i!=SIZE ; i++)
            h_vector[i] = h_vector[i]*h_vector[i];
        __STOP__(hRun)
    }
    showMean(cRun);
    showMean(ckRun);
    showMean(hRun);
}    

