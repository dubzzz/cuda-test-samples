#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <vector>

#define MAX_THREADS 256
#define SIZE 131072

#define __START__ hipEventCreate(&start); hipEventCreate(&stop); hipEventRecord(start, 0);
#define __STOP__(_V) hipEventRecord(stop, 0); hipEventSynchronize(stop); hipEventElapsedTime(&time, start, stop); _V.push_back(time); hipEventDestroy(start); hipEventDestroy(stop);
#define __NEXT__(_V) __STOP__(_V) __START__

template<class T>
__global__ void square_kernel(T *d_vector)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= SIZE)
        return;
    d_vector[i] = d_vector[i]*d_vector[i];
}

void showMean(std::vector<float> v)
{
    float sum(0);
    for (unsigned int i(0) ; i!=v.size() ; i++)
        sum += v[i];
    std::cout << 1000.*sum/v.size() << " microseconds" << std::endl;
}

int main(int argc, char **argv)
{
    std::cout << "SIZE (Datatype): " << SIZE << std::endl;
    hipEvent_t start, stop;
    std::vector<float> intRun, floatRun, doubleRun;
    float time(0);
    
    hipFree(0); // Force runtime API context establishment
    int h_vector_i[SIZE]; // For input and output
    float h_vector_f[SIZE]; // For input and output
    double h_vector_d[SIZE]; // For input and output
    for (unsigned int i(0) ; i!=SIZE ; i++)
    {
        h_vector_i[i] = i;
        h_vector_f[i] = i;
        h_vector_d[i] = i;
    }
    
    for (unsigned int i(0) ; i!=1000 ; i++)
    {
        int *d_vector_i;
        float *d_vector_f;
        double *d_vector_d;
        
        /* INT */
        hipMalloc(&d_vector_i, SIZE*sizeof(int));
        hipMemcpy(d_vector_i, h_vector_i, SIZE*sizeof(int), hipMemcpyHostToDevice);
        __START__
        square_kernel<<<(SIZE+MAX_THREADS-1)/MAX_THREADS, MAX_THREADS>>>(d_vector_i);
        hipDeviceSynchronize(); // Block until the device is finished
        __STOP__(intRun);
        hipMemcpy(h_vector_i, d_vector_i, SIZE*sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_vector_i);
        
        /* FLOAT */
        hipMalloc(&d_vector_f, SIZE*sizeof(float));
        hipMemcpy(d_vector_f, h_vector_f, SIZE*sizeof(float), hipMemcpyHostToDevice);
        __START__
        square_kernel<<<(SIZE+MAX_THREADS-1)/MAX_THREADS, MAX_THREADS>>>(d_vector_f);
        hipDeviceSynchronize(); // Block until the device is finished
        __STOP__(floatRun);
        hipMemcpy(h_vector_f, d_vector_f, SIZE*sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_vector_f);
        
        /* DOUBLE */
        hipMalloc(&d_vector_d, SIZE*sizeof(double));
        hipMemcpy(d_vector_d, h_vector_d, SIZE*sizeof(double), hipMemcpyHostToDevice);
        __START__
        square_kernel<<<(SIZE+MAX_THREADS-1)/MAX_THREADS, MAX_THREADS>>>(d_vector_d);
        hipDeviceSynchronize(); // Block until the device is finished
        __STOP__(doubleRun);
        hipMemcpy(h_vector_d, d_vector_d, SIZE*sizeof(double), hipMemcpyDeviceToHost);
        hipFree(d_vector_d);
    }
    showMean(intRun);
    showMean(floatRun);
    showMean(doubleRun);
}    

